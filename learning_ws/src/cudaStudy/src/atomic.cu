#include <hip/hip_runtime.h>

#include <iostream>
#include <stdio.h>
#include <string.h>

/*
    对同一块内存进行操作时，需要用到原子操作，不然就是乱序的
*/

__global__ void AtomicAdd(int* d_in)
{
    int Idx = blockDim.x * blockIdx.x + threadIdx.x;

    atomicAdd(&d_in[Idx], 1);

    // d_in[Idx] += 1;
}

int main(int argc, char** argv)
{

    const int arrSize = 1024;
    const int arrBytes = arrSize * sizeof(int);

    int h_in[arrSize];
    int h_out[arrSize];
    int* d_in = nullptr;

    for (int i = 0; i < arrSize; i++) {
        h_in[i] = i;
    }

    hipMalloc((void**)&d_in, arrBytes); // 注意这里取地址

    hipMemcpy(d_in, h_in, arrBytes, hipMemcpyHostToDevice);

    AtomicAdd<<<10, 100>>>(d_in);

    hipMemcpy(h_out, d_in, arrBytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < arrSize; i++) {
        printf("h_out[%d]:%d\n", i, h_out[i]);
    }

    hipFree(d_in);

    hipDeviceReset();
    return 0;
}